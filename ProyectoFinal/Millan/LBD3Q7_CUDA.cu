#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>
#include <omp.h>
#include "Random64.h"
#include <sys/stat.h>   // Para la función mkdir
#include <sys/types.h>  // Para el tipo de datos mode_t


const double deltax=0.1;//metro por celda
const double Lx_real=19.7;
const double Ly_real=26.5;
const double Lz_real=8;
const int Lx=Lx_real/deltax+2;
const int Ly=Ly_real/deltax+2;
const int Lz=Lz_real/deltax+2;
const double deltaT=0.50000*deltax/300.0;//segundo por click 



const int Q=7;
const double W0=1.0/4;

const double C=0.5;
const double C2=C*C;
const double AUX0=1-4*C2*(1-W0);

const double tau=0.5;
const double Utau=1.0/tau;
const double UmUtau= 1-Utau;

const double D = 0.987;




//-------------------------------------------------Clase LatticeBoltzman-------------------------------------------------
class LatticeBoltzman{
private:
    double w[Q];      //Pesos
    int Vx[Q], Vy[Q], Vz[Q]; //Vectores velocidad
    double *f, *fnew; //Funciones de distribucion
public:
    LatticeBoltzman(void);
    ~LatticeBoltzman(void);
    int n(int ix, int iy, int iz, int i){return (ix*Ly*Lz+iy*Lz+iz)*Q+i;};
    double rho(int ix, int iy, int iz, bool UseNew);
    double Jx(int ix, int iy, int iz, bool UseNew);
    double Jy(int ix, int iy, int iz, bool UseNew);
    double Jz(int ix, int iy, int iz, bool UseNew);
    double feq(double rho0, double Jx0, double Jy0, double Jz0, int i);
    void Inicie(double rho0, double Jx0, double Jy0, double Jz0);
    void Colision(void);
    void ImponerCampos(int t);
    void Adveccion(void);
    void Print(const char * NameFile,int z);
    friend class Fuentes; // Declarar a la clase Fuentes como amiga
};
LatticeBoltzman::LatticeBoltzman(void){
    //Cargar los pesos
    w[0]=W0; w[1]=w[2]=w[3]=w[4]=w[5]=w[6]=1.0/8;
    //Cargar los vectores velocidad
    Vx[0]=0; Vx[1]=1; Vx[2]=-1; Vx[3]=0; Vx[4]=0;  Vx[5]=0; Vx[6]=0;
    Vy[0]=0; Vy[1]=0; Vy[2]=0;  Vy[3]=1; Vy[4]=-1; Vy[5]=0; Vy[6]=0;
    Vz[0]=0; Vz[1]=0; Vz[2]=0;  Vz[3]=0; Vz[4]=0;  Vz[5]=1; Vz[6]=-1;
    //Crear los arreglos dinamicos
    int ArraySize=Lx*Ly*Lz*Q;
    f=new double [ArraySize]; fnew=new double [ArraySize];
}
LatticeBoltzman::~LatticeBoltzman(void){
    delete[] f; delete[] fnew;
}
double LatticeBoltzman::rho(int ix, int iy, int iz, bool UseNew){
    double sum; int i, n0;
    for(sum=0, i=0;i<Q;i++){
        n0=n(ix,iy,iz,i);
        if(UseNew) sum+=fnew[n0]; else sum+=f[n0];
    }
    return sum;
}
double LatticeBoltzman::Jx(int ix, int iy, int iz, bool UseNew){
    double sum; int i, n0;
    for(sum=0, i=0;i<Q;i++){
        n0=n(ix,iy,iz,i);
        if(UseNew) sum+=Vx[i]*fnew[n0]; else sum+=Vx[i]*f[n0];
    }
    return sum;
}
double LatticeBoltzman::Jy(int ix, int iy, int iz, bool UseNew){
    double sum; int i, n0;
    for(sum=0, i=0;i<Q;i++){
        n0=n(ix,iy,iz,i);
        if(UseNew) sum+=Vy[i]*fnew[n0]; else sum+=Vy[i]*f[n0];
    }
    return sum;
}
double LatticeBoltzman::Jz(int ix, int iy, int iz, bool UseNew){
    double sum; int i, n0;
    for(sum=0, i=0;i<Q;i++){
        n0=n(ix,iy,iz,i);
        if(UseNew) sum+=Vz[i]*fnew[n0]; else sum+=Vz[i]*f[n0];
    }
    return sum;
}
double LatticeBoltzman::feq(double rho0, double Jx0, double Jy0, double Jz0, int i){
    if(i>0)
        return 4*w[i]*(C2*rho0+Vx[i]*Jx0+Vy[i]*Jy0+Vz[i]*Jz0);
    else
        return rho0*AUX0;

}
void LatticeBoltzman::Inicie(double rho0, double Jx0, double Jy0, double Jz0){
    int ix, iy, iz, i, n0;
    for(ix=0;ix<Lx;ix++)      //Para cada celda
        for(iy=0;iy<Ly;iy++)
            for(iz=0;iz<Lz;iz++)
                for(i=0;i<Q;i++){ //En cada direccion
                    n0=n(ix,iy,iz,i);
                    f[n0]=feq(rho0,Jx0,Jy0,Jz0,i);
                }  
}
__global__ void ColisionKernel(double *f, double *fnew, int Lx, int Ly, int Lz, double Utau, double UmUtau, double D) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix < Lx && iy < Ly && iz < Lz) {
        int index = (ix * Ly * Lz + iy * Lz + iz) * 7;
        double rho0 = 0.0, Jx0 = 0.0, Jy0 = 0.0, Jz0 = 0.0;
        for (int i = 0; i < 7; i++) {
            rho0 += f[index + i];  // Calcula la densidad
        }
        // Aplicar las condiciones y actualizaciones (ejemplo)
        if (ix == Lx - 2 || ix == 1 || iy == Ly - 2 || iy == 1 || iz == Lz - 2 || iz == 1) {
            fnew[index + 0] = D * f[index + 0];
            fnew[index + 1] = D * f[index + 2];
            // Completar el resto de las asignaciones
        } else if (ix == Lx - 1 || ix == 0 || iy == Ly - 1 || iy == 0 || iz == Lz - 1 || iz == 0) {
            for (int i = 0; i < 7; i++) {
                fnew[index + i] = 0.0;
            }
        } else {
            for (int i = 0; i < 7; i++) {
                fnew[index + i] = UmUtau * f[index + i] + Utau * /* feq(...) */;
            }
        }
    }
}
void LatticeBoltzman::Colision(void) {
    double *d_f, *d_fnew;

    // Tamaño de los arreglos
    int size = Lx * Ly * Lz * Q * sizeof(double);

    // Reservar memoria en la GPU
    hipMalloc((void**)&d_f, size);
    hipMalloc((void**)&d_fnew, size);

    // Copiar datos desde la CPU a la GPU
    hipMemcpy(d_f, f, size, hipMemcpyHostToDevice);
    hipMemcpy(d_fnew, fnew, size, hipMemcpyHostToDevice);

    // Configuración de la cuadrícula y bloques
    dim3 threadsPerBlock(8, 8, 8);  // 512 hilos por bloque
    dim3 numBlocks((Lx + 7) / 8, (Ly + 7) / 8, (Lz + 7) / 8);  // Dividir el trabajo en bloques

    // Llamar al kernel
    ColisionKernel<<<numBlocks, threadsPerBlock>>>(d_f, d_fnew, Lx, Ly, Lz, Utau, UmUtau, D);

    // Sincronizar para asegurar que el kernel ha terminado
    hipDeviceSynchronize();

    // Copiar los resultados de vuelta a la CPU
    hipMemcpy(fnew, d_fnew, size, hipMemcpyDeviceToHost);

    // Liberar la memoria de la GPU
    hipFree(d_f);
    hipFree(d_fnew);
}
__global__ void AdveccionKernel(double *f, double *fnew, int *Vx, int *Vy, int *Vz, int Lx, int Ly, int Lz) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix < Lx && iy < Ly && iz < Lz) {
        for (int i = 0; i < 7; i++) {
            int ixnext = (ix + Vx[i] + Lx) % Lx;
            int iynext = (iy + Vy[i] + Ly) % Ly;
            int iznext = (iz + Vz[i] + Lz) % Lz;
            int n0 = (ix * Ly * Lz + iy * Lz + iz) * 7 + i;
            int n0next = (ixnext * Ly * Lz + iynext * Lz + iznext) * 7 + i;
            f[n0next] = fnew[n0];
        }
    }
}


void LatticeBoltzman::ImponerCampos(int t){}
void LatticeBoltzman::Adveccion(void) {
    double *d_f, *d_fnew;
    int *d_Vx, *d_Vy, *d_Vz;

    // Tamaño de los arreglos
    int size = Lx * Ly * Lz * Q * sizeof(double);
    int Vsize = Q * sizeof(int);

    // Reservar memoria en la GPU
    hipMalloc((void**)&d_f, size);
    hipMalloc((void**)&d_fnew, size);
    hipMalloc((void**)&d_Vx, Vsize);
    hipMalloc((void**)&d_Vy, Vsize);
    hipMalloc((void**)&d_Vz, Vsize);

    // Copiar los datos desde la CPU a la GPU
    hipMemcpy(d_f, f, size, hipMemcpyHostToDevice);
    hipMemcpy(d_fnew, fnew, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Vx, Vx, Vsize, hipMemcpyHostToDevice);
    hipMemcpy(d_Vy, Vy, Vsize, hipMemcpyHostToDevice);
    hipMemcpy(d_Vz, Vz, Vsize, hipMemcpyHostToDevice);

    // Configurar la cuadrícula y los bloques
    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((Lx + 7) / 8, (Ly + 7) / 8, (Lz + 7) / 8);

    // Llamar al kernel
    AdveccionKernel<<<numBlocks, threadsPerBlock>>>(d_f, d_fnew, d_Vx, d_Vy, d_Vz, Lx, Ly, Lz);

    // Sincronizar para asegurarse de que el kernel ha terminado
    hipDeviceSynchronize();

    // Copiar los resultados de vuelta a la CPU
    hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost);

    // Liberar memoria de la GPU
    hipFree(d_f);
    hipFree(d_fnew);
    hipFree(d_Vx);
    hipFree(d_Vy);
    hipFree(d_Vz);
}

void LatticeBoltzman::Print(const char * NameFile,int z){
    std::ofstream MyFile(NameFile); double rho0; int ix, iy;
    int iz = z;
    for(ix=0;ix<Lx;ix++){
        for(iy=0;iy<Ly;iy++){
            rho0=rho(ix,iy,iz,true);
            MyFile<<(float)ix*deltax<<" "<<(float)iy*deltax<<" "<<rho0<<std::endl;
        }
        MyFile<<std::endl;
    }
    MyFile.close();
}
//-------------------------------------------------Clase Fuentes-------------------------------------------------
class Fuentes {
private:
    std::string archivotxt;
    int ix, iy, iz;
    std::vector<double> sonido;
    LatticeBoltzman &LB;
public:
    Fuentes(std::string nombreArchivo, LatticeBoltzman& LBn, int Ix, int Iy, int Iz, int tmax)
        : archivotxt(nombreArchivo), ix(Ix), iy(Iy), iz(Iz), LB(LBn), sonido(int(tmax/deltaT),0) {
        std::ifstream archivo(nombreArchivo);
        if (!archivo.is_open()) {
            std::cerr << "Error al abrir el archivo: " << nombreArchivo << std::endl;
            return;
        }
        double valor;
        int i = 0;
        while (archivo >> valor && i < tmax) {
            sonido[i] = 20e-6*std::pow(10,valor/20.0);
            i++;
        }
        archivo.close();
        // std::cout << "Archivo " << nombreArchivo << " leído con éxito. Valores almacenados en el vector 'sonido'." << std::endl;
        // std::cout << "Contenido del vector 'sonido':" << std::endl;
        // for (int i = 0; i < tmax; i++) {
        //     std::cout << "sonido[" << i << "] = " << sonido[i] << std::endl;
        // }

    }
    void ImponerFuente(int t);
    friend class LatticeBoltzman;
};



void Fuentes::ImponerFuente(int t) {
    double rho0 =sonido[t];//amplitud * sin(omega * t);
    double Jx0 = LB.Jx(ix, iy, iz, false);
    double Jy0 = LB.Jy(ix, iy, iz, false);
    double Jz0 = LB.Jz(ix, iy, iz, false);
    for (int i = 0; i < Q; i++) {
        int n0 = LB.n(ix, iy, iz, i);
        LB.fnew[n0] = LB.feq(rho0, Jx0, Jy0, Jz0, i);
    }
}

//-------------------------------------------------Función principal-------------------------------------------------

int main(void){


    // Establecer el número de hilos de forma explícita
    int num_threads = 4;
    omp_set_num_threads(num_threads);
    LatticeBoltzman Ondas;
    int t;
    double tmax=20.0;//segundos 
    double rho0=0.0, Jx0=0, Jy0=0, Jz0=0;

    // Imprimir la cantidad de celdas en cada eje y los valores de deltax y deltaT
    std::cout << "Cantidad de celdas en el eje X (Lx): " << Lx << std::endl;
    std::cout << "Cantidad de celdas en el eje Y (Ly): " << Ly << std::endl;
    std::cout << "Cantidad de celdas en el eje Z (Lz): " << Lz << std::endl;
    std::cout << "Valor de deltax: " << deltax << " metros por celda" << std::endl;
    std::cout << "Valor de deltaT: " << deltaT << " segundos por click" << std::endl;
    std::cout << "Valor de tiempo total: " << tmax << " segundos "<<tmax/deltaT<<" click"<< std::endl;


    //INICIE

    Ondas.Inicie(rho0,Jx0,Jy0,Jz0);
    
    //Corre
    // ...

    auto start = std::chrono::high_resolution_clock::now(); // Start timer
    // Fuentes
    Crandom ran64(23);
    const int Numero_fuentes=5;
    Fuentes* fuentes[Numero_fuentes];
    int random_number_x;
    int random_number_y;
    int txt_number;
    
    for(int r=0;r<Numero_fuentes;r++){// Get a random integer between 0 and 120
        random_number_x= ran64.intRange(Lx/4,Lx*3/4);
        random_number_y= ran64.intRange(Lx/4,Ly*3/4);
        txt_number=ran64.intRange(1,5);
        fuentes[r] = new Fuentes("Fuentes/fuente_" + std::to_string(txt_number) + ".txt", Ondas, Lx/2, Ly/2, Lz/2,tmax); 
    }
    
    for(t=0; t<int(tmax/deltaT); t++){
        Ondas.Colision();
        Ondas.ImponerCampos(t);
        for(int r=0; r<Numero_fuentes; r++) {
            fuentes[r]->ImponerFuente(t);
        }
        Ondas.Adveccion();
        if(t % int(0.4/deltaT) == 0){
            std::cout << "Imprimendo: " << t << " click "<<(double)t*deltaT<<" segundos"<< std::endl;
            #pragma omp parallel
            for(int z=Lz/4; z<Lz; z=z+Lz/4){
                // Crear la carpeta D3/z si no existe
                char directory[30];
                sprintf(directory, "D3/%d", z);

                // Verificar si el directorio existe, si no, crearlo
                struct stat st = {0};
                if (stat(directory, &st) == -1) {
                    mkdir(directory, 0700);  // Crear el directorio con permisos de lectura/escritura
                }

                // Crear el archivo y guardar los datos
                char filename[50];
                sprintf(filename, "D3/%d/Ondas_%d.txt", z, int(1000*t*deltaT));
                Ondas.Print(filename, z);
            }
        }
        std::clog << t*deltaT << "     " << Ondas.rho(Lz/2, Ly/2, Lz/2, true) << std::endl;
    }


    auto end = std::chrono::high_resolution_clock::now(); // End timer
    auto duration = std::chrono::duration_cast<std::chrono::seconds>(end - start); // Calculate duration in milliseconds
    std::cout << "Total time for all iterations: " << duration.count() << " seconds" << std::endl;
    //Print
    
    return 0;
}